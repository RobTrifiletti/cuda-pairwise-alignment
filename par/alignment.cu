#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "alignment.h"
#include "scorematrix.h"

#define BLOCK_SIZE_X 1
#define BLOCK_SIZE_Y 1024

// Must be an even number
// Largest possible: 7776
#define COMPUTE_SIZE 6000

#define INNER 0
#define OUTER 1
#define TOP 2

#define max(a, b) (a > b ? a : b)

__device__ __constant__ int s[SCORE_MATRIX_SIZE][SCORE_MATRIX_SIZE];
__device__ __constant__ int gap;

__device__ __shared__ int diags[2][BLOCK_SIZE_Y + 1];
__device__ __shared__ int top[COMPUTE_SIZE + 1];

__device__ __shared__ int result_thread;

__device__ __shared__ char s_A[BLOCK_SIZE_Y];
__device__ __shared__ char s_B[BLOCK_SIZE_Y + COMPUTE_SIZE];

__device__ int mmax(int a, int b, int c, int d) {
  return max(a, max(b, max(c, d)));
}

__global__ void
__launch_bounds__ (1024)
alignment_kernel(int *T, char *A, char *B, int t_M, int t_N, int k, int T_middle) {
  int x = -1;
  int y = -1;
  int t = k & 1;

  // Set x, y if we are in a root thread.
  for (int j = 0; j <= k / 2; j++) {
    int tmp_x = 2 * j + t;
    int tmp_y = k / 2 - j;
    if (blockIdx.x == tmp_x && blockIdx.y == tmp_y &&
        tmp_x <= t_N / COMPUTE_SIZE + 1 && tmp_y <= t_M / BLOCK_SIZE_Y) {
      x = tmp_x;
      y = tmp_y;
      break;
    }
  }
  if (x == -1 && y == -1) {
    return;
  }

  result_thread = INT_MIN;

  int ty_mod_2 = threadIdx.y & 1;
  int ty_div_2 = threadIdx.y >> 1;
  int BS_div_2 = BLOCK_SIZE_Y >> 1;
  
  int col = x * COMPUTE_SIZE - threadIdx.y - (COMPUTE_SIZE - (BLOCK_SIZE_Y - 1));
  int row = y * BLOCK_SIZE_Y + threadIdx.y;

  int num_of_blocks = k / 2 + 1;
  int block_num = num_of_blocks - y - 1;

  int data_size = (COMPUTE_SIZE + 2 * BLOCK_SIZE_Y);

  int vertical_offset = - k / 2 * BLOCK_SIZE_Y;
  int first_block_offset = - (COMPUTE_SIZE + BLOCK_SIZE_Y - 1);
  int odd_block_offset = t * COMPUTE_SIZE;
  int block_num_offset = (data_size + COMPUTE_SIZE - BLOCK_SIZE_Y) * block_num;

  int first_write_idx = T_middle + vertical_offset + first_block_offset + odd_block_offset + block_num_offset;
  int second_write_idx = first_write_idx + COMPUTE_SIZE;
  int third_write_idx = second_write_idx + BLOCK_SIZE_Y;
  
  int first_read_idx = first_write_idx - 1;
  int second_read_idx = first_read_idx + BLOCK_SIZE_Y;
  int third_read_idx = second_read_idx + BLOCK_SIZE_Y;

  // Read the relevant part of the two sequences into shared memory. 
  if (row > 0 && row <= t_M) {
    s_A[threadIdx.y] = A[row - 1];    
  }

  for (int i = 0; i <= (COMPUTE_SIZE + 1) / BLOCK_SIZE_Y + 1; i++) {
    if (i * BLOCK_SIZE_Y + col > 0 && i * BLOCK_SIZE_Y + col <= t_N &&
  i * BLOCK_SIZE_Y + (BLOCK_SIZE_Y - threadIdx.y - 1) < COMPUTE_SIZE + BLOCK_SIZE_Y) {
      s_B[i * BLOCK_SIZE_Y + (BLOCK_SIZE_Y - threadIdx.y - 1)] = B[i * BLOCK_SIZE_Y + col - 1];
    }
  }

  // Read into INNER and OUTER in shared memory
  int t_col = x * COMPUTE_SIZE + ty_div_2 - COMPUTE_SIZE - 1;
  if (t_col >= 0) {
    diags[ty_mod_2][ty_div_2 + ty_mod_2] = T[first_read_idx + threadIdx.y];
  } else {
    diags[ty_mod_2][ty_div_2 + ty_mod_2] = INT_MIN;
  }

  int t_row = y * BLOCK_SIZE_Y + BS_div_2 - (threadIdx.y + 1) / 2 - 1;
  if (t_col + BS_div_2 >= 0 && t_row >= 0) {
    diags[ty_mod_2][(ty_div_2) + BS_div_2 + ty_mod_2] = T[second_read_idx + threadIdx.y];
  } else {
    diags[ty_mod_2][(ty_div_2) + BS_div_2 + ty_mod_2] = INT_MIN;
  }
  

  // Read into TOP in shared memory
  for (int i = 0; i < (COMPUTE_SIZE + 1) / BLOCK_SIZE_Y + 1; i++) {
    if (i * BLOCK_SIZE_Y + threadIdx.y < COMPUTE_SIZE + 1) {
      int top_idx = col + i * BLOCK_SIZE_Y + 2 * threadIdx.y;
      if (y > 0 && top_idx >= 0) {
  top[i * BLOCK_SIZE_Y + threadIdx.y] = T[third_read_idx + i * BLOCK_SIZE_Y + threadIdx.y];
      } else {
  top[i * BLOCK_SIZE_Y + threadIdx.y] = INT_MIN;
      }
    }
  }

  int limit;
  if (row == t_M - 1 && col + COMPUTE_SIZE - 1 > t_N - 1) {
    result_thread = threadIdx.y;
    limit = t_N - col;
  } else {
    limit = COMPUTE_SIZE;
  }

  // Make sure that all three diags are written and every thread sees result_thread.
  __syncthreads();

  int fill_diag = 0;
  int v1;
  int v2;
  int v3;
  int v4;

  int bs_minus_ty = BLOCK_SIZE_Y - threadIdx.y;
  int bs_minus_ty_minus_one = BLOCK_SIZE_Y - threadIdx.y - 1;
  char s_A_value = s_A[threadIdx.y] & 3;
 
  int res = diags[fill_diag][bs_minus_ty_minus_one];
  
  // Start computation
  for (int i = 0; i < limit; i++) {
    diags[fill_diag][BLOCK_SIZE_Y] = top[i];

    // Diagonal
    v1 = diags[1 - fill_diag][bs_minus_ty] + s[ s_A_value ][ s_B[i + bs_minus_ty_minus_one] & 3 ];

    // Above
    v2 = diags[fill_diag][bs_minus_ty] + gap;

    // To the left
    v3 = res + gap;

    // (0,0) is guaranteed to be "small" negative number this way. 99 is to ensure a large value in the score matrix to not give a positive yield.
    v4 = - row - abs(col + i) * 99;

    fill_diag = 1 - fill_diag;
    
    res = mmax(v1, v2, v3, v4);
    diags[fill_diag][bs_minus_ty_minus_one] = res;

    // Sync before reading from fill_diag.
    __syncthreads();
    top[i] = diags[fill_diag][0];
  }
  
  if (threadIdx.y == 0) {
    diags[fill_diag][BLOCK_SIZE_Y] = top[COMPUTE_SIZE];
  }
  __syncthreads();
  
  // Write TOP array
  int t_row_top = y * BLOCK_SIZE_Y + BLOCK_SIZE_Y - 1;
  for (int i = 0; i < COMPUTE_SIZE / BLOCK_SIZE_Y + 1; i++) {
    int t_col_top = i * BLOCK_SIZE_Y + col - BLOCK_SIZE_Y + 2 * threadIdx.y + 1;
    if (t_row_top >= 0 && t_row_top < t_M && t_col_top >= 0 && t_col_top < t_N && i * BLOCK_SIZE_Y + threadIdx.y < COMPUTE_SIZE) {
      T[first_write_idx + i * BLOCK_SIZE_Y + threadIdx.y] = top[i * BLOCK_SIZE_Y + threadIdx.y];
    }
  }
  
  // Write the first half of INNER/OUTER array
  int t_row_first = row - threadIdx.y + BLOCK_SIZE_Y - ty_div_2 - 2;
  int t_col_first = col + COMPUTE_SIZE - BLOCK_SIZE_Y + threadIdx.y + ty_div_2 + ty_mod_2;
  if (t_row_first >= 0 && t_row_first < t_M && t_col_first >= 0 && t_col_first < t_N) {
    T[second_write_idx + threadIdx.y] = diags[1 - (ty_mod_2)][ty_div_2 + 1];
  }

  // Write the second half of INNER/OUTER array
  int t_row_second = t_row_first - BS_div_2;
  int t_col_second = t_col_first + BS_div_2;
  if (t_row_second >= 0 && t_row_second < t_M && t_col_second >= 0 && t_col_second < t_N) {
    T[third_write_idx + threadIdx.y] = diags[1 - (ty_mod_2)][(ty_div_2) + 1 + BS_div_2];
  }

  // The result is written to the start of T
  if (result_thread >= 0) {
    T[0] = diags[fill_diag][BLOCK_SIZE_Y - result_thread - 1];
  }

  // Wait for global memory writes to finish
  __syncthreads();
  __threadfence();
}

void alignment(char *h_A, char *h_B, int M, int N, ScoreMatrix *sm) {
  // Translate characters to score matrix indexes
  for (int i = 0; i < M; i++) {
    switch (h_A[i]) {
    case 'a': h_A[i] = 0; break;
    case 'c': h_A[i] = 1; break;
    case 'g': h_A[i] = 2; break;
    case 't': h_A[i] = 3; break;
    case 'A': h_A[i] = 0; break;
    case 'C': h_A[i] = 1; break;
    case 'G': h_A[i] = 2; break;
    case 'T': h_A[i] = 3; break;
    }
  }
  for (int i = 0; i < N; i++) {
    switch (h_B[i]) {
    case 'a': h_B[i] = 0; break;
    case 'c': h_B[i] = 1; break;
    case 'g': h_B[i] = 2; break;
    case 't': h_B[i] = 3; break;
    case 'A': h_B[i] = 0; break;
    case 'C': h_B[i] = 1; break;
    case 'G': h_B[i] = 2; break;
    case 'T': h_B[i] = 3; break;
    }
  }

  // Copy A and B to Device Memory.
  char *d_A = copy_string_to_device(h_A, M);
  char *d_B = copy_string_to_device(h_B, N);

  // The table is (M + 1) x (N + 1)
  int t_M = M + 1;
  int t_N = N + 1;

  // Setup dimensions of grid/blocks.
  dim3 blockDim(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
  int gridX = (int) ceil(t_N / (double) COMPUTE_SIZE) + 1;
  int gridY = (int) ceil(t_M / (double) BLOCK_SIZE_Y);
  dim3 gridDim(gridX, gridY, 1);

  // Iterate over the score matrix and copy it into standard two-dim array. 
  int tmp_s[SCORE_MATRIX_SIZE][SCORE_MATRIX_SIZE];
  for(int i = 0; i < SCORE_MATRIX_SIZE; i++){
    for(int j = 0; j < SCORE_MATRIX_SIZE; j++){
      tmp_s[i][j] = sm->matrix[i][j];
    }
  }

  // Copy the score matrix and gap cost to constant memory on GPU
  hipMemcpyToSymbol(HIP_SYMBOL(s), tmp_s, 
    SCORE_MATRIX_SIZE * SCORE_MATRIX_SIZE * sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(gap), &sm->gap, sizeof(int));

  // Calculate the number of kernels to be invoked

  int row = 2 * t_M / BLOCK_SIZE_Y * BLOCK_SIZE_Y - t_M - 1;
  int col = BLOCK_SIZE_Y - 1 - row;
  int extra;
  if (col > t_N - 1) {
    extra = 0;
  } else {
    extra = 1;
  }
  int limit = ceil(2 * t_M / (float) BLOCK_SIZE_Y) + extra +
   floor((t_N / (float) COMPUTE_SIZE));
  
  // Allocate global memory array for partial results
  int *d_T;
  int d_T_size = (t_N / COMPUTE_SIZE + 1) * (2 * (t_M / BLOCK_SIZE_Y + 1) + COMPUTE_SIZE + 1) * sizeof(int) * 2;
  hipMalloc(&d_T, d_T_size);

  // Calculate the center cell in the arrays for partial results
  int T_middle = d_T_size / sizeof(int) / 2 - 1;

  // Invoke kernel
  for (int k = 0; k < limit; k++) {
    alignment_kernel<<< gridDim, blockDim >>>(d_T, d_A, d_B, t_M, t_N, k, T_middle);
  }
  
  // Copy result to host
  int res;
  hipMemcpy(&res, d_T, sizeof(int), hipMemcpyDeviceToHost);

  // Print optimal score
  printf("Optimal score: %i\n", res);

  // Free memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_T);
}

char *copy_string_to_device(char *h_str, int len) {
  char *d_str;
  hipMalloc(&d_str, len);
  hipMemcpy(d_str, h_str, len, hipMemcpyHostToDevice);
  return d_str;
}

int *allocate_device_table(int M, int N) {
  int *table;
  hipMalloc(&table, M * N * sizeof(int));
  hipMemset(table, 0, M * N * sizeof(int));
  return table;
}
